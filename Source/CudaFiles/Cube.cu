#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include "cutil_math.h"
#include "../Render/CUDATables.h"
#include "../Render/Defines.h"

//-- Table textures
texture<unsigned int, 1, hipReadModeElementType> tEdgeTex;
texture<unsigned int, 1, hipReadModeElementType> tNRVertsTex;
texture<unsigned int, 1, hipReadModeElementType> tTriTex;

//-- Sampling data
texture<unsigned char, 1, hipReadModeNormalizedFloat> tData1;
texture<unsigned char, 1, hipReadModeNormalizedFloat> tData2; //-- octave 2
texture<unsigned char, 1, hipReadModeNormalizedFloat> tData3; //-- octave 3


//-- load tables into 
extern "C"
void load_tables_as_textures(unsigned int** aEdgeTable, unsigned int** aTriTable, unsigned int** aNrVertsTable)
{
	// set up format
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);

	// allocate tables as textures
    cutilSafeCall(hipMalloc((void**) aEdgeTable, 256*sizeof(unsigned int)));
    cutilSafeCall(hipMemcpy((void*)* aEdgeTable, (void *)edgeTable, 256*sizeof(unsigned int), hipMemcpyHostToDevice) );
    cutilSafeCall(hipBindTexture(0, tEdgeTex, *aEdgeTable, channelDesc) );

    cutilSafeCall(hipMalloc((void**) aNrVertsTable, 256*sizeof(unsigned int)));
    cutilSafeCall(hipMemcpy((void*)* aNrVertsTable, (void *)numVertsTable, 256*sizeof(unsigned int), hipMemcpyHostToDevice) );
    cutilSafeCall(hipBindTexture(0, tNRVertsTex, *aNrVertsTable, channelDesc) );

    cutilSafeCall(hipMalloc((void**) aTriTable, 256*16*sizeof(unsigned int)));
    cutilSafeCall(hipMemcpy((void*)*aTriTable, (void *)triTable, 256*16*sizeof(unsigned int), hipMemcpyHostToDevice) );
    cutilSafeCall(hipBindTexture(0, tTriTex, *aTriTable, channelDesc) );
}

extern "C"
void load_perlin_data(float** aDstCuda1, float* aData1, float** aDstCuda2, float* aData2, float** aDstCuda3, float* aData3, unsigned int dataSize)
{
	hipChannelFormatDesc cD = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	// allocate data as textures
    cutilSafeCall(hipMalloc((void**) aDstCuda1, dataSize*sizeof(float)));
    cutilSafeCall(hipMemcpy((void*)* aDstCuda1, (void *)aData1, dataSize*sizeof(float), hipMemcpyHostToDevice) );
	cutilSafeCall(hipBindTexture(0, tData1, *aDstCuda1, cD) );

    cutilSafeCall(hipMalloc((void**) aDstCuda2, dataSize*sizeof(float)));
    cutilSafeCall(hipMemcpy((void*)* aDstCuda2, (void *)aData2, dataSize*sizeof(float), hipMemcpyHostToDevice) );
    cutilSafeCall(hipBindTexture(0, tData2, *aDstCuda2, cD) );

    cutilSafeCall(hipMalloc((void**) aDstCuda3, dataSize*sizeof(float)));
    cutilSafeCall(hipMemcpy((void*)* aDstCuda3, (void *)aData3, dataSize*sizeof(float), hipMemcpyHostToDevice) );
    cutilSafeCall(hipBindTexture(0, tData3, *aDstCuda2, cD) );
}

__device__ float SampleData1( uint3 pos )
{
    unsigned int i = (pos.z*PERLIN_DATA_RANK*PERLIN_DATA_RANK) + (pos.y*PERLIN_DATA_RANK) + pos.x;
    return tex1Dfetch(tData1, i);
}

__device__ float SampleData2( uint3 pos )
{
	unsigned int i = (pos.z*PERLIN_DATA_RANK*PERLIN_DATA_RANK) + (pos.y*PERLIN_DATA_RANK) + pos.x;
    return tex1Dfetch(tData2, i);
}

__device__ float SampleData3( uint3 pos )
{
    unsigned int i = (pos.z*PERLIN_DATA_RANK*PERLIN_DATA_RANK) + (pos.y*PERLIN_DATA_RANK) + pos.x;
    return tex1Dfetch(tData3, i);
}


__global__ void cuda_CreateCube(float3* aVertList, float3* aNormList, unsigned int* aIndexList)
{
	int column = ( blockDim.x * blockIdx.x) + threadIdx.x;
	int row = ( blockDim.y * blockIdx.y) + threadIdx.y;
	int depth = ( blockDim.z * blockIdx.z) + threadIdx.z;

	int vertex = (column + (row * blockDim.x * gridDim.x)) + (depth * blockDim.x * gridDim.x * blockDim.y * gridDim.y);
	vertex *= 15;
	int triangle = vertex;

	//float xdim = (float)column * 0.05f;
	//float ydim = (float)row * 0.05f;
	//float zdim = (float)depth * 0.05f;

	{
	float points[8];
	unsigned int bitmap = 0;

		points[0] = SampleData1( make_uint3( row, column, depth ) );
		points[1] = SampleData1( make_uint3( row, column+1, depth ) );
		points[2] = SampleData1( make_uint3( row+1, column+1, depth ) );
		points[3] = SampleData1( make_uint3( row+1, column, depth ) );
		points[4] = SampleData1( make_uint3( row, column, depth+1 ) );
		points[5] = SampleData1( make_uint3( row, column+1, depth+1 ) );
		points[6] = SampleData1( make_uint3( row+1, column+1, depth+1 ) );
		points[7] = SampleData1( make_uint3( row+1, column, depth+1 ) );

		//--Create lookup bitmap to find the edge table
		for (int i = 0; i < 8; i++)
		{
			if (points[i] > 0)
			{
				bitmap ^= 1<<i;						
			}
		}

		//-- Creating triangles
		unsigned int edge = tex1Dfetch(tEdgeTex, bitmap);
		unsigned int vertNr = tex1Dfetch(tNRVertsTex ,bitmap);
		if (vertNr >= 255 || vertNr <= 0)
		{

		}
		else
		{
			float3 vertsPos[12]; 
			//-- front
			vertsPos[0]  = make_float3(0,		0.5,	0);
			vertsPos[1]  = make_float3(0.5f,	1,		0);
			vertsPos[2]  = make_float3(1,		0.5f,	0);
			vertsPos[3]  = make_float3(0.5f,	0,		0);

			//-- back
			vertsPos[4]  = make_float3(0,		0.5f,	1);
			vertsPos[5]  = make_float3(0.5f,	1,		1);
			vertsPos[6]  = make_float3(1,		0.5f,	1);
			vertsPos[7]  = make_float3(0.5f,	0,		1);

			//-- mid
			vertsPos[8]  = make_float3(0,		0,		0.5f);
			vertsPos[9]  = make_float3(0,		1,		0.5f);
			vertsPos[10]  = make_float3(1,		1,		0.5f);
			vertsPos[11]  = make_float3(1,		0,		0.5f);
		

			for (int i = 0; i < vertNr; i++)
			{
				aVertList[vertex] = vertsPos[tex1Dfetch(tTriTex, (bitmap * 16) + i)];
				aNormList[vertex] = make_float3(0,0,0);
				aVertList[vertex] += make_float3( row, column, depth);
				aVertList[vertex] *= make_float3( 0.05, 0.05, 0.05);
				aIndexList[triangle] = vertex++;
				triangle++;
				

				//aVertList[vertex] = vertsPos[triTablePC[bitmap][i]];
				//aVertList[vertex] += make_float3( row, column, depth );
				//aTriList[triangle] = vertex++;
				//triangle++;
				//
				//aVertList[vertex] = vertsPos[triTablePC[bitmap][i]];
				//aVertList[vertex] += make_float3( row, column, depth );
				//aTriList[triangle] = vertex++;
				//triangle++;
			}

		}


	}	
}

__global__ void generateNormals(float3* aVertList, float3* aNormList, unsigned int* aIndexList)
{
	
}


extern "C"
void launch_CreateCube(dim3 grid, dim3 threads, float3* aVertList, float3* aNormList, unsigned int* aIndexList)
{
	//Here you normally run:
	cuda_CreateCube<<<grid, threads>>>(aVertList, aNormList, aIndexList);
	cutilCheckMsg("cuda_CreateCube failed");
	
}



//--the cuber
//__global__ void cuda_CreateCube(float3* aVertList, float3* aNormList, unsigned int* aIndexList)
//{
//	int column = ( blockDim.x * blockIdx.x) + threadIdx.x;
//	int row = ( blockDim.y * blockIdx.y) + threadIdx.y;
//	int depth = ( blockDim.z * blockIdx.z) + threadIdx.z;
//
//	int vertex = (column + (row * blockDim.x * gridDim.x)) + (depth * blockDim.x * gridDim.x * blockDim.y * gridDim.y);
//	vertex *= 3;
//	int triangle = vertex;
//
//	if( vertex < 500000)
//	{
//		float xdim = (float)column * 0.05;
//		float ydim = (float)row * 0.05;
//		float zdim = (float)depth * 0.05;
//		aVertList[vertex] = make_float3(0.01 + xdim,ydim, zdim);
//		aVertList[vertex+1] = make_float3(xdim, 0.01 + ydim, zdim);
//		aVertList[vertex+2] = make_float3(xdim, ydim, zdim);
//		
//		aNormList[vertex] = make_float3(0,0,1);
//		aNormList[vertex+1] = make_float3(0,0,1);
//		aNormList[vertex+2] = make_float3(0,0,1);
//		
//		aIndexList[triangle] = vertex;
//		aIndexList[triangle+1] = vertex+1;
//		aIndexList[triangle+2] = vertex+2;
//	}
//}