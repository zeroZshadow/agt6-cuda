#include "hip/hip_runtime.h"
#include <cutil_inline.h>

__global__ void cuda_CreateCube(float3* aVertList, float3* aNormList, unsigned int* aIndexList)
{
	//aVertList, aNormList and aIndexList are simple arrays with the same size as the VBO's
	//You can fill them like normal and they will be rendered propperly
}

extern "C"
void launch_CreateCube(dim3 grid, dim3 threads, float3* aVertList, float3* aNormList, unsigned int* aIndexList)
{
	//Here you normally run:
	cuda_CreateCube<<<grid, threads>>>(aVertList, aNormList, aIndexList);
	cutilCheckMsg("cuda_CreateCube failed");
	
	//TEST
	
	aVertList[0] = make_float3(1,0,0);
	aVertList[1] = make_float3(0,1,0);
	aVertList[2] = make_float3(0,0,0);
	
	aNormList[0] = make_float3(0,0,1);
	aNormList[1] = make_float3(0,0,1);
	aNormList[2] = make_float3(0,0,1);
	
	aIndexList[0] = 0;
	aIndexList[1] = 1;
	aIndexList[2] = 2;
}