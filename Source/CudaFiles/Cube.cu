#include "hip/hip_runtime.h"
#include <cutil_inline.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include "cutil_math.h"
#include "../Render/CUDATables.h"
#include "../Render/Defines.h"

//-- Table textures
texture<unsigned int, 1, hipReadModeElementType> tEdgeTex;
texture<unsigned int, 1, hipReadModeElementType> tNRVertsTex;
texture<unsigned int, 1, hipReadModeElementType> tTriTex;

//-- Sampling data
texture<unsigned char, 1, hipReadModeNormalizedFloat> tData1;
texture<unsigned char, 1, hipReadModeNormalizedFloat> tData2; //-- octave 2
texture<unsigned char, 1, hipReadModeNormalizedFloat> tData3; //-- octave 3



//-- load tables into 
extern "C"
void load_tables_as_textures(unsigned int** aEdgeTable, unsigned int** aTriTable, unsigned int** aNrVertsTable)
{
	// set up format
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);

	// allocate tables as textures
    cutilSafeCall(hipMalloc((void**) aEdgeTable, 256*sizeof(unsigned int)));
    cutilSafeCall(hipMemcpy((void*)* aEdgeTable, (void *)edgeTable, 256*sizeof(unsigned int), hipMemcpyHostToDevice) );
    cutilSafeCall(hipBindTexture(0, tEdgeTex, *aEdgeTable, channelDesc) );

    cutilSafeCall(hipMalloc((void**) aNrVertsTable, 256*sizeof(unsigned int)));
    cutilSafeCall(hipMemcpy((void*)* aNrVertsTable, (void *)numVertsTable, 256*sizeof(unsigned int), hipMemcpyHostToDevice) );
    cutilSafeCall(hipBindTexture(0, tNRVertsTex, *aNrVertsTable, channelDesc) );

    cutilSafeCall(hipMalloc((void**) aTriTable, 256*16*sizeof(unsigned int)));
    cutilSafeCall(hipMemcpy((void*)*aTriTable, (void *)triTable, 256*16*sizeof(unsigned int), hipMemcpyHostToDevice) );
    cutilSafeCall(hipBindTexture(0, tTriTex, *aTriTable, channelDesc) );
}

extern "C"
void load_perlin_data(float** aDstCuda1, float* aData1, float** aDstCuda2, float* aData2, float** aDstCuda3, float* aData3, unsigned int dataSize)
{
	hipChannelFormatDesc cD = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

	// allocate data as textures
    cutilSafeCall(hipMalloc((void**) aDstCuda1, dataSize*sizeof(float)));
    cutilSafeCall(hipMemcpy((void*)* aDstCuda1, (void *)aData1, dataSize*sizeof(float), hipMemcpyHostToDevice) );
	cutilSafeCall(hipBindTexture(0, tData1, *aDstCuda1, cD) );

    cutilSafeCall(hipMalloc((void**) aDstCuda2, dataSize*sizeof(float)));
    cutilSafeCall(hipMemcpy((void*)* aDstCuda2, (void *)aData2, dataSize*sizeof(float), hipMemcpyHostToDevice) );
    cutilSafeCall(hipBindTexture(0, tData2, *aDstCuda2, cD) );

    cutilSafeCall(hipMalloc((void**) aDstCuda3, dataSize*sizeof(float)));
    cutilSafeCall(hipMemcpy((void*)* aDstCuda3, (void *)aData3, dataSize*sizeof(float), hipMemcpyHostToDevice) );
    cutilSafeCall(hipBindTexture(0, tData3, *aDstCuda2, cD) );
}

__device__ float SampleData1( int3 pos )
{
	//pos.x += 7;
	//pos.y += 7;
	//pos.z += 7;
	pos.x %= PERLIN_DATA_RANK;
	pos.y %= PERLIN_DATA_RANK;
	pos.z %= PERLIN_DATA_RANK;
    unsigned int i = (pos.z*PERLIN_DATA_RANK*PERLIN_DATA_RANK) + (pos.y*PERLIN_DATA_RANK) + pos.x;
    return tex1Dfetch(tData1, i);
}

__device__ float SampleData2( int3 pos )
{
	//pos.x += 11;
	//pos.y += 11;
	//pos.z += 11;	
	pos.x %= PERLIN_DATA_RANK;
	pos.y %= PERLIN_DATA_RANK;
	pos.z %= PERLIN_DATA_RANK;
	unsigned int i = (pos.z*PERLIN_DATA_RANK*PERLIN_DATA_RANK) + (pos.y*PERLIN_DATA_RANK) + pos.x;
    return tex1Dfetch(tData2, i);
}

__device__ float SampleData3( int3 pos )
{
	pos.x %= PERLIN_DATA_RANK;
	pos.y %= PERLIN_DATA_RANK;
	pos.z %= PERLIN_DATA_RANK;
    unsigned int i = (pos.z*PERLIN_DATA_RANK*PERLIN_DATA_RANK) + (pos.y*PERLIN_DATA_RANK) + pos.x;
    return tex1Dfetch(tData3, i);
}


__device__ float DensityWithFloor( int3 pos, float floorMultiplier)
{
	return SampleData1( pos ) + SampleData2( pos ) * 0.1f + SampleData3( pos ) * 0.05f - (pos.y * floorMultiplier);
}


__device__
float3 InterpVertexPos(float3 p0, float3 p1, float f0, float f1)
{
    float t = clamp(f0 / (f0 - f1), 0.0f , 1.0f);
	return lerp(p0, p1, t);
} 

__device__ 
float3 InterpVertexPos2(float3 p0, float3 p1, float f0, float f1)
{
    float t = clamp(f1 / (f1 - f0), 0.0f , 1.0f);
	return lerp(p0, p1, t);
}

__global__ void cuda_CreateCube(float3 pos, float3* aVertList, float3* aNormList, unsigned int* aIndexList)
{
	int column = ( blockDim.x * blockIdx.x) + threadIdx.x;
	int row = ( blockDim.y * blockIdx.y) + threadIdx.y;
	int depth = ( blockDim.z * blockIdx.z) + threadIdx.z;

	int vertex = (column + (row * blockDim.x * gridDim.x)) + (depth * blockDim.x * gridDim.x * blockDim.y * gridDim.y);
	vertex *= 15;
	int triangle = vertex;

	int x = column + pos.x * MARCHING_BLOCK_SIZE;
	int y = row + pos.y * MARCHING_BLOCK_SIZE;
	int z = depth + pos.z * MARCHING_BLOCK_SIZE;

	float points[8];
	int bitmap = 0;

	points[0] = DensityWithFloor( make_int3( x, y, z ),  0.03f );
	points[1] = DensityWithFloor( make_int3( x+1, y, z ), 0.03f );
	points[2] = DensityWithFloor( make_int3( x+1, y+1, z ), 0.03f);
	points[3] = DensityWithFloor( make_int3( x, y+1, z ), 0.03f );
	points[4] = DensityWithFloor( make_int3( x, y, z+1 ), 0.03f);
	points[5] = DensityWithFloor( make_int3( x+1, y, z+1) , 0.03f);
	points[6] = DensityWithFloor( make_int3( x+1, y+1, z+1 ),0.03f);;
	points[7] = DensityWithFloor( make_int3( x, y+1, z+1 ), 0.03f);	

	for(int i = 0; i < 15; i++)
	{
		aVertList[vertex + i] = make_float3(0); 
		aNormList[vertex + i] = make_float3(0);
		aIndexList[triangle+ i] = 0;
	}


	//--Create lookup bitmap to find the edge table
	for (int i = 0; i < 8; i++)
	{
		//points[i] -= column * 0.1;
		if (points[i] > 0)
		{
			bitmap ^= 1<<i;						
		}
	}

	//-- Creating triangles
	unsigned int edge = tex1Dfetch(tEdgeTex, bitmap);
	unsigned int vertNr = tex1Dfetch(tNRVertsTex ,bitmap);
	if (vertNr >= 255 || vertNr <= 0)
	{
		
	}
	else
	{
		//-- point cube
		float3 pCube[8];
		pCube[0] = make_float3(0, 0, 0);
		pCube[1] = make_float3(1, 0, 0);
		pCube[2] = make_float3(1, 1, 0);
		pCube[3] = make_float3(0, 1, 0);
		pCube[4] = make_float3(0, 0, 1);
		pCube[5] = make_float3(1, 0, 1);
		pCube[6] = make_float3(1, 1, 1);
		pCube[7] = make_float3(0, 1, 1);
		

	float3 vertlist[12];

    vertlist[0] = InterpVertexPos( pCube[0], pCube[1], points[0], points[1]);
    vertlist[1] = InterpVertexPos( pCube[1], pCube[2], points[1], points[2]);
    vertlist[2] = InterpVertexPos( pCube[2], pCube[3], points[2], points[3]);
    vertlist[3] = InterpVertexPos( pCube[3], pCube[0], points[3], points[0]);

	vertlist[4] = InterpVertexPos( pCube[4], pCube[5], points[4], points[5]);
    vertlist[5] = InterpVertexPos( pCube[5], pCube[6], points[5], points[6]);
    vertlist[6] = InterpVertexPos( pCube[6], pCube[7], points[6], points[7]);
    vertlist[7] = InterpVertexPos( pCube[7], pCube[4], points[7], points[4]);

	vertlist[8] = InterpVertexPos( pCube[0], pCube[4], points[0], points[4]);
    vertlist[9] = InterpVertexPos( pCube[1], pCube[5], points[1], points[5]);
    vertlist[10] = InterpVertexPos( pCube[2], pCube[6], points[2], points[6]);
    vertlist[11] = InterpVertexPos( pCube[3], pCube[7], points[3], points[7]);

		for (int i = 0; i < vertNr; i+=3)
		{
			int dst = vertex + i;
			aVertList[dst] = vertlist[tex1Dfetch(tTriTex, (bitmap * 16) + i +2)];
			aVertList[dst] += make_float3(x,y,z);
			aIndexList[dst] = dst;

			aVertList[dst+1] = vertlist[tex1Dfetch(tTriTex, (bitmap * 16) + i + 1)];
			aVertList[dst+1] += make_float3(x,y,z);
			aIndexList[dst+1] = dst+1;

			aVertList[dst+2] = vertlist[tex1Dfetch(tTriTex, (bitmap * 16) + i + 0)];
			aVertList[dst+2] += make_float3(x,y,z);
			aIndexList[dst+2] = dst+2;

		}
	}

}

__global__ void cuda_generateNormals(float3* aVertList, float3* aNormList, unsigned int* aIndexList)
{
	int column = ( blockDim.x * blockIdx.x) + threadIdx.x;
	int row = ( blockDim.y * blockIdx.y) + threadIdx.y;
	int depth = ( blockDim.z * blockIdx.z) + threadIdx.z;

	int vertex = (column + (row * blockDim.x * gridDim.x)) + (depth * blockDim.x * gridDim.x * blockDim.y * gridDim.y);
	vertex *= 15;
	int triangle = vertex;

	//-- Generate normals
	float3 vec1, vec2, vec3;
	float3 normal;
	
	for (int i = vertex; i < vertex+15; i+=3)
	{	
		if(aIndexList[i] == 0 && aIndexList[i+1] == 0 && aIndexList[i+2] ==0)
		{continue;}

		vec1 = aVertList[i];
		vec2 = aVertList[i+1];
		vec3 = aVertList[i+2];

		float3 edge1 = vec2 - vec1;
		float3 edge2 = vec3 - vec1;
		normal = cross(edge1, edge2);
		normal = normalize(normal);

		aNormList[i] = normal;
		aNormList[i+1] = normal;
		aNormList[i+2] = normal;
	}
}


extern "C"
void launch_CreateCube(dim3 grid, dim3 threads, float3 aPos, float3* aVertList, float3* aNormList, unsigned int* aIndexList)
{
	//Here you normally run:
	
	cuda_CreateCube<<<grid, threads>>>( aPos, aVertList, aNormList, aIndexList);
	cutilCheckMsg("cuda_CreateCube failed");
	cuda_generateNormals<<<grid, threads>>>(aVertList, aNormList, aIndexList);
	cutilCheckMsg("cuda_CreateCube failed");
	

}




//--the cuber
//__global__ void cuda_CreateCube(float3* aVertList, float3* aNormList, unsigned int* aIndexList)
//{
//	int column = ( blockDim.x * blockIdx.x) + threadIdx.x;
//	int row = ( blockDim.y * blockIdx.y) + threadIdx.y;
//	int depth = ( blockDim.z * blockIdx.z) + threadIdx.z;
//
//	int vertex = (column + (row * blockDim.x * gridDim.x)) + (depth * blockDim.x * gridDim.x * blockDim.y * gridDim.y);
//	vertex *= 3;
//	int triangle = vertex;
//
//	if( vertex < 500000)
//	{
//		float xdim = (float)column * 0.05;
//		float ydim = (float)row * 0.05;
//		float zdim = (float)depth * 0.05;
//		aVertList[vertex] = make_float3(0.01 + xdim,ydim, zdim);
//		aVertList[vertex+1] = make_float3(xdim, 0.01 + ydim, zdim);
//		aVertList[vertex+2] = make_float3(xdim, ydim, zdim);
//		
//		aNormList[vertex] = make_float3(0,0,1);
//		aNormList[vertex+1] = make_float3(0,0,1);
//		aNormList[vertex+2] = make_float3(0,0,1);
//		
//		aIndexList[triangle] = vertex;
//		aIndexList[triangle+1] = vertex+1;
//		aIndexList[triangle+2] = vertex+2;
//	}
//}


/////////////////////////////First cuda cube marching block
/////////////////////////////
//__global__ void cuda_CreateCube(float3 pos, float3* aVertList, float3* aNormList, unsigned int* aIndexList)
//{
//	int column = ( blockDim.x * blockIdx.x) + threadIdx.x;
//	int row = ( blockDim.y * blockIdx.y) + threadIdx.y;
//	int depth = ( blockDim.z * blockIdx.z) + threadIdx.z;
//
//	int vertex = (column + (row * blockDim.x * gridDim.x)) + (depth * blockDim.x * gridDim.x * blockDim.y * gridDim.y);
//	vertex *= 15;
//	int triangle = vertex;
//
//	int x = column + pos.x;
//	int y = row + pos.y;
//	int z = depth + pos.z;
//
//	float points[8];
//	unsigned int bitmap = 0;
//
//	points[0] = DensityWithFloor( make_uint3( y, x, z ),  0.05f );
//	points[1] = DensityWithFloor( make_uint3( y, x+1, z ), 0.05f );
//	points[2] = DensityWithFloor( make_uint3( y+1, x+1, z ), 0.05f);
//	points[3] = DensityWithFloor( make_uint3( y+1, x, z ), 0.05f );
//	points[4] = DensityWithFloor( make_uint3( y, x, z+1 ), 0.05f);
//	points[5] = DensityWithFloor( make_uint3( y, x+1, z+1) , 0.05f);
//	points[6] = DensityWithFloor( make_uint3( y+1, x+1, z+1 ),0.05f);
//	points[7] = DensityWithFloor( make_uint3( y+1, x, z+1 ), 0.05f);
//
//	for(int i = 0; i < 15; i++)
//	{
//		aVertList[vertex + i] = make_float3(0); 
//		aNormList[vertex + i] = make_float3(0);
//		aIndexList[triangle+ i] = 0;
//	}
//
//
//	//--Create lookup bitmap to find the edge table
//	for (int i = 0; i < 8; i++)
//	{
//		//points[i] -= column * 0.1;
//		if (points[i] > 0)
//		{
//			bitmap ^= 1<<i;						
//		}
//	}
//
//	//-- Creating triangles
//	unsigned int edge = tex1Dfetch(tEdgeTex, bitmap);
//	unsigned int vertNr = tex1Dfetch(tNRVertsTex ,bitmap);
//	if (vertNr >= 255 || vertNr <= 0)
//	{
//
//	}
//	else
//	{
//		float3 vertsPos[12]; 
//		//-- front
//		vertsPos[0]  = make_float3(0,		0.5,	0);
//		vertsPos[1]  = make_float3(0.5f,	1,		0);
//		vertsPos[2]  = make_float3(1,		0.5f,	0);
//		vertsPos[3]  = make_float3(0.5f,	0,		0);
//
//		//-- back
//		vertsPos[4]  = make_float3(0,		0.5f,	1);
//		vertsPos[5]  = make_float3(0.5f,	1,		1);
//		vertsPos[6]  = make_float3(1,		0.5f,	1);
//		vertsPos[7]  = make_float3(0.5f,	0,		1);
//
//		//-- mid
//		vertsPos[8]  = make_float3(0,		0,		0.5f);
//		vertsPos[9]  = make_float3(0,		1,		0.5f);
//		vertsPos[10]  = make_float3(1,		1,		0.5f);
//		vertsPos[11]  = make_float3(1,		0,		0.5f);
//	
//
//		for (int i = 0; i < vertNr; i++)
//		{
//			aVertList[vertex] = vertsPos[tex1Dfetch(tTriTex, (bitmap * 16) + i)];
//			aVertList[vertex] += make_float3( row, column, depth);
//			//aVertList[vertex] *= make_float3( 0.05, 0.05, 0.05);
//			aIndexList[triangle] = vertex++;
//			triangle++;
//		}
//	}
//
//}