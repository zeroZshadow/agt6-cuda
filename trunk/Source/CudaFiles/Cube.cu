#include "hip/hip_runtime.h"
////---CUda marching. 
////* compacting voxel code is taken from 

#include <cutil_inline.h>
#include <hip/hip_runtime_api.h>
#include <thrust/device_vector.h>
#include <thrust/scan.h>
#include "cutil_math.h"
#include "../Render/CUDATables.h"
#include "../Render/Defines.h"

// Data
#define SIZE 256
#define MASK 0xFF

//Perlin Tables GPU
unsigned char* cuda_p;
float* cuda_gx;
float* cuda_gy;
float* cuda_gz;
texture<unsigned char> tP;
texture<float> tGX;
texture<float> tGY;
texture<float> tGZ;

//-- Table textures
texture<unsigned int, 1, hipReadModeElementType> tEdgeTex;
texture<unsigned int, 1, hipReadModeElementType> tNRVertsTex;
texture<unsigned int, 1, hipReadModeElementType> tTriTex;

//-- Sampling data
float* dataPerlin1;
float* dataPerlin2;
float* dataPerlin3;
int rankPerlin;
texture<unsigned char, 1, hipReadModeNormalizedFloat> tPerlin1;
texture<unsigned char, 1, hipReadModeNormalizedFloat> tPerlin2; //-- octave 2
texture<unsigned char, 1, hipReadModeNormalizedFloat> tPerlin3; //-- octave 3

extern __device__ float Noise3(float x, float y, float z);

//-- load tables into 
extern "C"
void load_tables_as_textures(unsigned int** aEdgeTable, unsigned int** aTriTable, unsigned int** aNrVertsTable)
{
	// set up format
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindUnsigned);

	// allocate tables as textures
    cutilSafeCall(hipMalloc((void**) aEdgeTable, 256*sizeof(unsigned int)));
    cutilSafeCall(hipMemcpy((void*)* aEdgeTable, (void *)edgeTable, 256*sizeof(unsigned int), hipMemcpyHostToDevice) );
    cutilSafeCall(hipBindTexture(0, tEdgeTex, *aEdgeTable, channelDesc) );

    cutilSafeCall(hipMalloc((void**) aNrVertsTable, 256*sizeof(unsigned int)));
    cutilSafeCall(hipMemcpy((void*)* aNrVertsTable, (void *)numVertsTable, 256*sizeof(unsigned int), hipMemcpyHostToDevice) );
    cutilSafeCall(hipBindTexture(0, tNRVertsTex, *aNrVertsTable, channelDesc) );

    cutilSafeCall(hipMalloc((void**) aTriTable, 256*16*sizeof(unsigned int)));
    cutilSafeCall(hipMemcpy((void*)*aTriTable, (void *)triTable, 256*16*sizeof(unsigned int), hipMemcpyHostToDevice) );
    cutilSafeCall(hipBindTexture(0, tTriTex, *aTriTable, channelDesc) );
}


__device__ float SampleData1( uint3 pos )
{
    unsigned int i = (pos.z*PERLIN_DATA_SIZE*PERLIN_DATA_SIZE) + (pos.y*PERLIN_DATA_SIZE) + pos.x;
    return tex1Dfetch(tPerlin1, i);
}

__device__ float SampleData2( uint3 pos )
{
	unsigned int i = (pos.z*PERLIN_DATA_SIZE*PERLIN_DATA_SIZE) + (pos.y*PERLIN_DATA_SIZE) + pos.x;
    return tex1Dfetch(tPerlin2, i);
}

__device__ float SampleData3( uint3 pos )
{
    unsigned int i = (pos.z*PERLIN_DATA_SIZE*PERLIN_DATA_SIZE) + (pos.y*PERLIN_DATA_SIZE) + pos.x;
    return tex1Dfetch(tPerlin3, i);
}

__device__ float DensityCaves( uint3 pos)
{
	return SampleData1( pos );
}

__device__ float DensityWithFloor( uint3 pos, float floor)
{
	return SampleData1( pos ) + (floor * 0.05f);
}

__device__ float DensitySphere( uint3 pos, float radius)
{
	float dens = length( make_float3(pos) - make_float3(32,32,32));
	dens = 3 - dens/9;
	return SampleData1( pos ) + dens;
}

__device__
float3 InterpVertexPos(float iso, float3 p0, float3 p1, float f0, float f1)
{
    float t = ((iso - f0) / (f1 - f0));//, 0.0f , 1.0f);
	return lerp(p0, p1, t);
} 

__device__ 
float3 InterpVertexPos2(float3 p0, float3 p1, float f0, float f1)
{
    float t = clamp(f1 / (f1 - f0), 0.0f , 1.0f);
	return lerp(p0, p1, t);
}

//-- Compacting code is partially from the NVidia examples
__global__ void cuda_ClassifyVoxel(GenerateInfo agInfo, float3 pos, uint* voxelVertCnt, 
								   uint* voxelOccupied)
{
	int3 gridPos;
	gridPos.x = ( blockDim.x * blockIdx.x) + threadIdx.x;
	gridPos.y = ( blockDim.y * blockIdx.y) + threadIdx.y;
	gridPos.z = ( blockDim.z * blockIdx.z) + threadIdx.z;
	float y = gridPos.y + pos.y * MARCHING_BLOCK_SIZE;
	int i = (gridPos.x + (gridPos.y * blockDim.x * gridDim.x)) + (gridPos.z * blockDim.x * gridDim.x * blockDim.y * gridDim.y);

	float points[8];
	int bitmap = 0;

	points[0] = DensityWithFloor( make_uint3( gridPos.x,	gridPos.y,	gridPos.z ),	0.03f) - ((float)y  * 0.03f) +1;
	points[1] = DensityWithFloor( make_uint3( gridPos.x+1,	gridPos.y,	gridPos.z ),	0.03f) - ((float)y * 0.03f) +1;
	points[2] = DensityWithFloor( make_uint3( gridPos.x+1,	gridPos.y+1,	gridPos.z ),	0.03f) - ((float)(y+1) * 0.03f)+1;
	points[3] = DensityWithFloor( make_uint3( gridPos.x,	gridPos.y+1,	gridPos.z ),	0.03f) - ((float)(y+1) * 0.03f)+1;
	points[4] = DensityWithFloor( make_uint3( gridPos.x,	gridPos.y,	gridPos.z+1 ),	0.03f) - ((float)y * 0.03f)+1;
	points[5] = DensityWithFloor( make_uint3( gridPos.x+1,	gridPos.y,	gridPos.z+1) ,	0.03f) - ((float)y * 0.03f)+1;
	points[6] = DensityWithFloor( make_uint3( gridPos.x+1,	gridPos.y+1,	gridPos.z+1 ),	0.03f) - ((float)(y+1) * 0.03f)+1;
	points[7] = DensityWithFloor( make_uint3( gridPos.x,	gridPos.y+1,	gridPos.z+1 ),	0.03f) - ((float)(y+1) * 0.03f)+1;

	for (int ii = 0; ii < 8; ii++)
	{
		if (points[ii] < agInfo.iso)
		{
			bitmap ^= 1<<ii;						
		}
	}

    // read number of vertices from texture
    uint numVerts = tex1Dfetch(tNRVertsTex, bitmap);
	voxelVertCnt[i] = numVerts;
    voxelOccupied[i] = (numVerts > 0);
}

extern "C" 
void launch_ClassifyVoxel( dim3 grid, dim3 threads, GenerateInfo agInfo, float3 pos, 
						  uint* voxelVertCnt, uint *voxelOccupied)
{
    // calculate number of vertices need per voxel
    cuda_ClassifyVoxel<<<grid, threads>>>(agInfo, pos, voxelVertCnt, voxelOccupied);
    cutilCheckMsg("classifyVoxel failed");
}

//// compact voxel array
//__global__ void
//cuda_CompactVoxels(uint *compactedVoxelArray, uint *voxelOccupied, uint *voxelOccupiedScan, uint numVoxels)
//{
//    uint blockId = __mul24(blockIdx.y, gridDim.x) + blockIdx.x;
//    uint i = __mul24(blockId, blockDim.x) + threadIdx.x;
//
//    if (voxelOccupied[i] && (i < numVoxels)) {
//        compactedVoxelArray[ voxelOccupiedScan[i] ] = i;
//    }
//}
//
//extern "C" void 
//launch_compactVoxels(dim3 grid, dim3 threads, uint *compactedVoxelArray, uint *voxelOccupied, uint *voxelOccupiedScan, uint numVoxels)
//{
//    compactVoxels<<<grid, threads>>>(compactedVoxelArray, voxelOccupied, 
//                                     voxelOccupiedScan, numVoxels);
//    cutilCheckMsg("compactVoxels failed");
//}


__device__ uint3 calcGridPos(uint i)
{
	uint3 gridPos;
	gridPos.x = i % MARCHING_BLOCK_SIZE;
	gridPos.y = i % MARCHING_BLOCK_SIZE;
	gridPos.z = i % MARCHING_BLOCK_SIZE;
}

__global__ void cuda_CreateCube(GenerateInfo agInfo, float3 pos, float3* aVertList, float3* aNormList, unsigned int* aIndexList)

{
	float iso = 0.9;
	int column = ( blockDim.x * blockIdx.x) + threadIdx.x;
	int row = ( blockDim.y * blockIdx.y) + threadIdx.y;
	int depth = ( blockDim.z * blockIdx.z) + threadIdx.z;

	int vertex = (column + (row * blockDim.x * gridDim.x)) + (depth * blockDim.x * gridDim.x * blockDim.y * gridDim.y);
	vertex *= 15;
	int triangle = vertex;

	int x = column + pos.x * MARCHING_BLOCK_SIZE;
	int y = row + pos.y * MARCHING_BLOCK_SIZE;
	int z = depth + pos.z * MARCHING_BLOCK_SIZE;

	for(int i = 0; i < 15; i++)
	{
		aVertList[vertex+i] += make_float3(0,0,0);
		aNormList[vertex+i] += make_float3(0,0,0);
		aIndexList[vertex+i] = 0;
	}

	float points[8];
	int bitmap = 0;

	points[0] = DensityWithFloor( make_uint3( column,	row,	depth ),	0.03f) - ((float)y  * 0.03f) +1;
	points[1] = DensityWithFloor( make_uint3( column+1,	row,	depth ),	0.03f) - ((float)y * 0.03f) +1;
	points[2] = DensityWithFloor( make_uint3( column+1,	row+1,	depth ),	0.03f) - ((float)(y+1) * 0.03f)+1;
	points[3] = DensityWithFloor( make_uint3( column,	row+1,	depth ),	0.03f) - ((float)(y+1) * 0.03f)+1;
	points[4] = DensityWithFloor( make_uint3( column,	row,	depth+1 ),	0.03f) - ((float)y * 0.03f)+1;
	points[5] = DensityWithFloor( make_uint3( column+1,	row,	depth+1) ,	0.03f) - ((float)y * 0.03f)+1;
	points[6] = DensityWithFloor( make_uint3( column+1,	row+1,	depth+1 ),	0.03f) - ((float)(y+1) * 0.03f)+1;
	points[7] = DensityWithFloor( make_uint3( column,	row+1,	depth+1 ),	0.03f) - ((float)(y+1) * 0.03f)+1;



	//--Create lookup bitmap to find the edge table
	for (int i = 0; i < 8; i++)
	{
		//points[i] -= column * 0.1;
		if (points[i] < iso)
		{
			bitmap ^= 1<<i;						
		}
	}

	//-- Creating triangles
	unsigned int vertNr = tex1Dfetch(tNRVertsTex ,bitmap);
	if (vertNr >= 255 || vertNr <= 0)
	{
		
	}
	else
	{
		//-- point cube
		float3 pCube[8];
		pCube[0] = make_float3(0, 0, 0);
		pCube[1] = make_float3(1, 0, 0);
		pCube[2] = make_float3(1, 1, 0);
		pCube[3] = make_float3(0, 1, 0);
		pCube[4] = make_float3(0, 0, 1);
		pCube[5] = make_float3(1, 0, 1);
		pCube[6] = make_float3(1, 1, 1);
		pCube[7] = make_float3(0, 1, 1);
		

		float3 vertlist[12];

		vertlist[0] = InterpVertexPos(iso, pCube[0], pCube[1], points[0], points[1]);
		vertlist[1] = InterpVertexPos(iso, pCube[1], pCube[2], points[1], points[2]);
		vertlist[2] = InterpVertexPos(iso, pCube[2], pCube[3], points[2], points[3]);
		vertlist[3] = InterpVertexPos(iso, pCube[3], pCube[0], points[3], points[0]);

		vertlist[4] = InterpVertexPos(iso, pCube[4], pCube[5], points[4], points[5]);
		vertlist[5] = InterpVertexPos(iso, pCube[5], pCube[6], points[5], points[6]);
		vertlist[6] = InterpVertexPos(iso, pCube[6], pCube[7], points[6], points[7]);
		vertlist[7] = InterpVertexPos(iso, pCube[7], pCube[4], points[7], points[4]);

		vertlist[8] = InterpVertexPos(iso, pCube[0], pCube[4], points[0], points[4]);
		vertlist[9] = InterpVertexPos(iso, pCube[1], pCube[5], points[1], points[5]);
		vertlist[10] = InterpVertexPos(iso, pCube[2], pCube[6], points[2], points[6]);
		vertlist[11] = InterpVertexPos(iso, pCube[3], pCube[7], points[3], points[7]);

		for (int i = 0; i < vertNr; i+=3)
		{
			int dst = vertex + i;
			aVertList[dst] = vertlist[tex1Dfetch(tTriTex, (bitmap * 16) + i +0)];
			aVertList[dst] += make_float3(x,y,z);
			aVertList[dst] *= make_float3(0.05,0.05,0.05);
			aIndexList[dst] = dst;

			aVertList[dst+1] = vertlist[tex1Dfetch(tTriTex, (bitmap * 16) + i + 1)];
			aVertList[dst+1] += make_float3(x,y,z);
			aVertList[dst+1] *= make_float3(0.05,0.05,0.05);
			aIndexList[dst+1] = dst+1;

			aVertList[dst+2] = vertlist[tex1Dfetch(tTriTex, (bitmap * 16) + i + 2)];
			aVertList[dst+2] += make_float3(x,y,z);
			aVertList[dst+2] *= make_float3(0.05,0.05,0.05);
			aIndexList[dst+2] = dst+2;

			float3 edge1 = aVertList[dst+1] - aVertList[dst];
			float3 edge2 = aVertList[dst+2] - aVertList[dst];
			aNormList[dst] = normalize(cross(edge1, edge2));
			aNormList[dst+1] = aNormList[dst];
			aNormList[dst+2] = aNormList[dst];			
		}
	}
}


extern "C"
void launch_CreateCube(GenerateInfo* agInfo, dim3 grid, dim3 threads, float3 aPos, float3* aVertList, float3* aNormList, unsigned int* aIndexList)
{
	//Here you normally run:	
	cuda_CreateCube<<<grid, threads>>>(*agInfo, aPos, aVertList, aNormList, aIndexList);
	cutilCheckMsg("cuda_CreateCube failed");
}

extern "C"
void host_InitPerlinData(int rank, int size)
{
	hipChannelFormatDesc cD = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	rankPerlin = rank;
	int dataSize = size*size*size;
	
	// allocate data as textures
    cutilSafeCall(hipMalloc((void**) &dataPerlin1, dataSize*sizeof(float)));
    cutilSafeCall(hipMalloc((void**) &dataPerlin2, dataSize*sizeof(float)));
    cutilSafeCall(hipMalloc((void**) &dataPerlin3, dataSize*sizeof(float)));
}



////
//This is old code used for reference!!!!!!!!!!!!!!!!!!!!!

//PERLIN

extern "C"
void host_PerlinInitialize(unsigned int nSeed)
{
	// Permutation table
	unsigned char p[SIZE];
	// Gradients
	float gx[SIZE];
	float gy[SIZE];
	float gz[SIZE];

	hipChannelFormatDesc cDUnsigned = hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
	hipChannelFormatDesc cDFloat = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	int i, j, nSwap;

	srand(nSeed);
	// Initialize the permutation table
	for(i = 0; i < SIZE; i++)
		p[i] = i;

	for(i = 0; i < SIZE; i++)
	{
		j = rand() & MASK;

		nSwap = p[i];
		p[i]  = p[j];
		p[j]  = nSwap;
	}

	// Generate the gradient lookup tables
	for(i = 0; i < SIZE; i++)
	{
		// Ken Perlin proposes that the gradients are taken from the unit 
		// circle/sphere for 2D/3D, but there are no noticable difference 
		// between that and what I'm doing here. For the sake of generality 
		// I will not do that.

		gx[i] = float(rand())/(RAND_MAX/2) - 1.0f; 
		gy[i] = float(rand())/(RAND_MAX/2) - 1.0f;
		gz[i] = float(rand())/(RAND_MAX/2) - 1.0f;
	}
	
	//Alloc Tables
	cutilSafeCall(hipMalloc((void**) &cuda_p, SIZE*sizeof(unsigned char)));
	cutilSafeCall(hipMalloc((void**) &cuda_gx, SIZE*sizeof(float)));
	cutilSafeCall(hipMalloc((void**) &cuda_gy, SIZE*sizeof(float)));
	cutilSafeCall(hipMalloc((void**) &cuda_gz, SIZE*sizeof(float)));
	
	cutilSafeCall(hipMemcpy((void*) cuda_p, (void *)p, SIZE*sizeof(unsigned char), hipMemcpyHostToDevice) );
	cutilSafeCall(hipMemcpy((void*) cuda_gx, (void *)gx, SIZE*sizeof(float), hipMemcpyHostToDevice) );
	cutilSafeCall(hipMemcpy((void*) cuda_gy, (void *)gy, SIZE*sizeof(float), hipMemcpyHostToDevice) );
	cutilSafeCall(hipMemcpy((void*) cuda_gz, (void *)gz, SIZE*sizeof(float), hipMemcpyHostToDevice) );
	
	cutilSafeCall(hipBindTexture(0, tP, cuda_p, cDUnsigned) );
	cutilSafeCall(hipBindTexture(0, tGX, cuda_gx, cDFloat) );
	cutilSafeCall(hipBindTexture(0, tGY, cuda_gy, cDFloat) );
	cutilSafeCall(hipBindTexture(0, tGZ, cuda_gz, cDFloat) );
}

__device__ float Noise1(float x)
{
		// Compute what gradients to use
	int qx0 = (int)floorf(x);
	int qx1 = qx0 + 1;
	float tx0 = x - (float)qx0;
	float tx1 = tx0 - 1;

	// Make sure we don't come outside the lookup table
	qx0 = qx0 & MASK;
	qx1 = qx1 & MASK;

	// Compute the dotproduct between the vectors and the gradients
	float v0 = tex1Dfetch(tGX, qx0)*tx0;
	float v1 = tex1Dfetch(tGX, qx1)*tx1;

	// Modulate with the weight function
	float wx = (3 - 2*tx0)*tx0*tx0;
	float v = v0 - wx*(v0 - v1);

	return v;
}

__device__ float Noise2(float x, float y)
{
	// Compute what gradients to use
	int qx0 = (int)floorf(x);
	int qx1 = qx0 + 1;
	float tx0 = x - (float)qx0;
	float tx1 = tx0 - 1;

	int qy0 = (int)floorf(y);
	int qy1 = qy0 + 1;
	float ty0 = y - (float)qy0;
	float ty1 = ty0 - 1;

	// Make sure we don't come outside the lookup table
	qx0 = qx0 & MASK;
	qx1 = qx1 & MASK;

	qy0 = qy0 & MASK;
	qy1 = qy1 & MASK;

	// Permutate values to get pseudo randomly chosen gradients
	int q00 = tex1Dfetch(tP, (qy0 + tex1Dfetch(tP, qx0)) & MASK);
	int q01 = tex1Dfetch(tP, (qy0 + tex1Dfetch(tP, qx1)) & MASK);

	int q10 = tex1Dfetch(tP, (qy1 + tex1Dfetch(tP, qx0)) & MASK);
	int q11 = tex1Dfetch(tP, (qy1 + tex1Dfetch(tP, qx1)) & MASK);

	// Compute the dotproduct between the vectors and the gradients
	float v00 = tex1Dfetch(tGX, q00)*tx0 + tex1Dfetch(tGY, q00) * ty0;
	float v01 = tex1Dfetch(tGX, q01)*tx1 + tex1Dfetch(tGY, q01) * ty0;

	float v10 = tex1Dfetch(tGX, q10)*tx0 + tex1Dfetch(tGY, q10) * ty1;
	float v11 = tex1Dfetch(tGX, q11)*tx1 + tex1Dfetch(tGY, q11) * ty1;

	// Modulate with the weight function
	float wx = (3 - 2*tx0)*tx0*tx0;
	float v0 = v00 - wx*(v00 - v01);
	float v1 = v10 - wx*(v10 - v11);

	float wy = (3 - 2*ty0)*ty0*ty0;
	float v = v0 - wy*(v0 - v1);

	return v;	
}

__device__ float Noise3(float x, float y, float z)
{
	// Compute what gradients to use
	int qx0 = (int)floorf(x);
	int qx1 = qx0 + 1;
	float tx0 = x - (float)qx0;
	float tx1 = tx0 - 1;

	int qy0 = (int)floorf(y);
	int qy1 = qy0 + 1;
	float ty0 = y - (float)qy0;
	float ty1 = ty0 - 1;

	int qz0 = (int)floorf(z);
	int qz1 = qz0 + 1;
	float tz0 = z - (float)qz0;
	float tz1 = tz0 - 1;

	// Make sure we don't come outside the lookup table
	qx0 = qx0 & MASK;
	qx1 = qx1 & MASK;

	qy0 = qy0 & MASK;
	qy1 = qy1 & MASK;

	qz0 = qz0 & MASK;
	qz1 = qz1 & MASK;

	// Permutate values to get pseudo randomly chosen gradients
	int q000 = tex1Dfetch(tP, (qz0 + tex1Dfetch(tP, (qy0 + tex1Dfetch(tP, qx0)) & MASK)) & MASK);
	int q001 = tex1Dfetch(tP, (qz0 + tex1Dfetch(tP, (qy0 + tex1Dfetch(tP, qx1)) & MASK)) & MASK);

	int q010 = tex1Dfetch(tP, (qz0 + tex1Dfetch(tP, (qy1 + tex1Dfetch(tP, qx0)) & MASK)) & MASK);
	int q011 = tex1Dfetch(tP, (qz0 + tex1Dfetch(tP, (qy1 + tex1Dfetch(tP, qx1)) & MASK)) & MASK);

	int q100 = tex1Dfetch(tP, (qz1 + tex1Dfetch(tP, (qy0 + tex1Dfetch(tP, qx0)) & MASK)) & MASK);
	int q101 = tex1Dfetch(tP, (qz1 + tex1Dfetch(tP, (qy0 + tex1Dfetch(tP, qx1)) & MASK)) & MASK);

	int q110 = tex1Dfetch(tP, (qz1 + tex1Dfetch(tP, (qy1 + tex1Dfetch(tP, qx0)) & MASK)) & MASK);
	int q111 = tex1Dfetch(tP, (qz1 + tex1Dfetch(tP, (qy1 + tex1Dfetch(tP, qx1)) & MASK)) & MASK);

	// Compute the dotproduct between the vectors and the gradients
	float v000 = tex1Dfetch(tGX, q000)*tx0 + tex1Dfetch(tGY, q000)*ty0 + tex1Dfetch(tGZ, q000)*tz0;
	float v001 = tex1Dfetch(tGX, q001)*tx1 + tex1Dfetch(tGY, q001)*ty0 + tex1Dfetch(tGZ, q001)*tz0;

	float v010 = tex1Dfetch(tGX, q010)*tx0 + tex1Dfetch(tGY, q010)*ty1 + tex1Dfetch(tGZ, q010)*tz0;
	float v011 = tex1Dfetch(tGX, q011)*tx1 + tex1Dfetch(tGY, q011)*ty1 + tex1Dfetch(tGZ, q011)*tz0;

	float v100 = tex1Dfetch(tGX, q100)*tx0 + tex1Dfetch(tGY, q100)*ty0 + tex1Dfetch(tGZ, q100)*tz1;
	float v101 = tex1Dfetch(tGX, q101)*tx1 + tex1Dfetch(tGY, q101)*ty0 + tex1Dfetch(tGZ, q101)*tz1;

	float v110 = tex1Dfetch(tGX, q110)*tx0 + tex1Dfetch(tGY, q110)*ty1 + tex1Dfetch(tGZ, q110)*tz1;
	float v111 = tex1Dfetch(tGX, q111)*tx1 + tex1Dfetch(tGY, q111)*ty1 + tex1Dfetch(tGZ, q111)*tz1;

	// Modulate with the weight function
	float wx = (3 - 2*tx0)*tx0*tx0;
	float v00 = v000 - wx*(v000 - v001);
	float v01 = v010 - wx*(v010 - v011);
	float v10 = v100 - wx*(v100 - v101);
	float v11 = v110 - wx*(v110 - v111);

	float wy = (3 - 2*ty0)*ty0*ty0;
	float v0 = v00 - wy*(v00 - v01);
	float v1 = v10 - wy*(v10 - v11);

	float wz = (3 - 2*tz0)*tz0*tz0;
	float v = v0 - wz*(v0 - v1);

	return v;	
}



//-- Taken from the NVidia cuda example
extern "C" void ThrustScanWrapper(unsigned int* output, unsigned int* input, unsigned int numElements)
{
    thrust::exclusive_scan(thrust::device_ptr<unsigned int>(input), 
                           thrust::device_ptr<unsigned int>(input + numElements),
                           thrust::device_ptr<unsigned int>(output));
}




__global__ void cuda_CreatePerlin(GenerateInfo agInfo, float3 pos, int rank, float* aPerlin1, float* aPerlin2, float* aPerlin3)
{
	#define PI				3.14159265358979323846264338327950288419716939937510582097494459072381640628620899862803482534211706798f
		float	piDev1				= PI * agInfo.prlnNoise1;
		float	piDev2				= PI * agInfo.prlnNoise2;
		float	piDev3				= PI * agInfo.prlnNoise3;
	#undef PI

	int column = ( blockDim.x * blockIdx.x) + threadIdx.x;
	int row = ( blockDim.y * blockIdx.y) + threadIdx.y;
	int depth = ( blockDim.z * blockIdx.z) + threadIdx.z;
	
	int voxel = (column + (row * blockDim.x * gridDim.x)) + (depth * blockDim.x * gridDim.x * blockDim.y * gridDim.y);
	float3 index = make_float3(column+(pos.x*MARCHING_BLOCK_SIZE), row+(pos.y*MARCHING_BLOCK_SIZE), depth+(pos.z*MARCHING_BLOCK_SIZE));
	
	aPerlin1[voxel] = Noise3(index.x / piDev1, index.y / piDev1, index.z / piDev1) * agInfo.prlnWeight1;
	aPerlin1[voxel] += Noise3((index.x+rank*2) / piDev2, index.y / piDev2, index.z / piDev2) * agInfo.prlnWeight2;
	aPerlin1[voxel] += Noise3((index.x+rank*4) / piDev3, index.y / piDev3, index.z / piDev3) * agInfo.prlnWeight3;
}


extern "C"
void host_CreatePerlinData(GenerateInfo* agInfo, dim3 grid, dim3 threads, float3 pos, int rank)
{
	hipChannelFormatDesc cD = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	
	//Unbind Textures
	cutilSafeCall(hipUnbindTexture(tPerlin1));
	cutilSafeCall(hipUnbindTexture(tPerlin2));
	cutilSafeCall(hipUnbindTexture(tPerlin3));
	
	//Fill perlins
	cuda_CreatePerlin<<<grid, threads>>>(*agInfo, pos, rank, dataPerlin1, dataPerlin2, dataPerlin3);
    cutilCheckMsg("cuda_CreateCube failed");

    //Bind as texture
	cutilSafeCall(hipBindTexture(0, tPerlin1, dataPerlin1, cD) );
	cutilSafeCall(hipBindTexture(0, tPerlin2, dataPerlin2, cD) );
	cutilSafeCall(hipBindTexture(0, tPerlin3, dataPerlin3, cD) );
}